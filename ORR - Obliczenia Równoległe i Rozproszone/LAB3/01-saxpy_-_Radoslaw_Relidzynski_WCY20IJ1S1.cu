#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 *
 * Some bugs have been placed in this codebase for your edification.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    c[i] = a[i] * 2 + b[i];
  }
}

int main()
{
    int deviceId;
    hipGetDevice(&deviceId);
    
    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // Initialize memory
    for( int i = 0; i < N; ++i )
    {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }

    // Send addresses for GPU
    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    // Get SM
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    int multiProcessorCount = props.multiProcessorCount;

    int threads_per_block = 512;
    // int number_of_blocks = (N / threads_per_block) + 1;
    int number_of_blocks = multiProcessorCount * 32;

    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    hipDeviceSynchronize();

    // Send addresses for CPU
    hipMemPrefetchAsync(c, size, hipCpuDeviceId);

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
