
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initWith(float num, float *xx, int N)
{
  // for(int i = 0; i < N; ++i)
  // {
  //   a[i] = num;
  // }
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  xx[gid] = num;
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  // for(int i = 0; i < N; ++i)
  // {
  //   result[i] = a[i] + b[i];
  // }
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  if (gid < N) {
    result[gid] = a[gid] + b[gid];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

/* async attempt
__global__ void checkElementsAre(float target, float *array, int N)
{
  // for(int i = 0; i < N; i++)
  // {
  //   if(array[i] != target)
  //   {
  //     printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
  //     exit(1);
  //   }
  // }
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  if(array[gid] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", gid, array[gid], target);
      // exit(1);  // an analogous instruction is needed to terminate GPU operations
    }
  
  // printf("SUCCESS! All values added correctly.\n");
}
*/


int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipError_t err;
  err = hipMallocManaged(&a, N);

  // a = (float *)malloc(size);
  // b = (float *)malloc(size);
  // c = (float *)malloc(size);

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  err = hipGetLastError();
  if (err != hipSuccess)                           // `cudaSuccess` is provided by CUDA.
  {
    printf("1 Error: %s\n", hipGetErrorString(err)); // `cudaGetErrorString` is provided by CUDA.
  }

  size_t threads_per_block = 1024;
  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;;

  // initWith(3, a, N);
  // initWith(4, b, N);
  // initWith(0, c, N);

  initWith<<<number_of_blocks, threads_per_block>>>(3, a, N);
  initWith<<<number_of_blocks, threads_per_block>>>(4, b, N);
  initWith<<<number_of_blocks, threads_per_block>>>(0, c, N);

  err = hipGetLastError();
  if (err != hipSuccess)                           // `cudaSuccess` is provided by CUDA.
  {
    printf("2 Error: %s\n", hipGetErrorString(err)); // `cudaGetErrorString` is provided by CUDA.
  }

  // addVectorsInto(c, a, b, N);
  addVectorsInto<<<number_of_blocks, threads_per_block>>>(c, a, b, N);
  hipDeviceSynchronize();

  err = hipGetLastError();
  if (err != hipSuccess)                           // `cudaSuccess` is provided by CUDA.
  {
    printf("3 Error: %s\n", hipGetErrorString(err)); // `cudaGetErrorString` is provided by CUDA.
  }

  checkElementsAre(7, c, N);

  /* async attempt
  checkElementsAre<<<number_of_blocks, threads_per_block>>>(7, c, N);
  printf("SUCCESS! All values added correctly.\n");
  */

  hipFree(a);
  hipFree(b);
  hipFree(c);

  err = hipGetLastError();
  if (err != hipSuccess)                           // `cudaSuccess` is provided by CUDA.
  {
    printf("4 Error: %s\n", hipGetErrorString(err)); // `cudaGetErrorString` is provided by CUDA.
  }
}
